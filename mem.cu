#include "hip/hip_runtime.h"
/**
 *  This is a tool to profile performance counters on GPU 
 * 
 *  It uses the CUDA Profiling Tools Interface (CUPTI) to
 *  periodically fetch GPU performance counters and store them
 *  in a file.
 * 
 *  The goal is to place this tool as a spy application that 
 *  montiors GPU usage of target application (eg - web browser)
 * 
*/

#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <cupti.h>
#include <time.h>

using namespace std;

////////////////////////////////////
//				CONFIG			  //
////////////////////////////////////

// Timer //
const int NUM_SECONDS = 1;
const int TIMES = 30;

// List of performance counters to fetch //
char *PC[] = {"sm_efficiency","achieved_occupancy","ipc","issued_ipc","issue_slot_utilization","gld_requested_throughput","gst_requested_throughput","gld_throughput","gst_throughput","tex_cache_throughput","tex_fu_utilization","single_precision_fu_utilization","stall_inst_fetch","stall_exec_dependency","stall_memory_dependency","stall_other","stall_constant_memory_dependency","stall_pipe_busy","stall_memory_throttle","stall_not_selected","l2_read_transactions","l2_tex_read_throughput","l2_tex_write_throughput","l2_read_throughput","l2_write_throughput","dram_utilization","eligible_warps_per_cycle"};
const char PC_COUNT = 27;

// Kernel launch //
const int N = 1024;
const int THREADS = 1024;
const int BLOCKS = 1;

////////////////////////////////////
//			GLOBAL VARS			  //
////////////////////////////////////
size_t f, t;
FILE *ptr;
char data[5000];
static uint64_t kernelDuration;
CUpti_SubscriberHandle subscriber[PC_COUNT];
hipCtx_t context = 0;
hipDevice_t device = 0;
int deviceNum = 0;
int deviceCount;
char deviceName[32];
CUpti_MetricID metricId[PC_COUNT];
CUpti_EventGroupSets *passData[PC_COUNT];
CUpti_MetricValue metricValue[PC_COUNT];

////////////////////////////////////
//				HELPERS			  //
////////////////////////////////////

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
	hipError_t _status = apiFuncCall;                                            \
	if (_status != hipSuccess) {                                             \
		fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
				__FILE__, __LINE__, #apiFuncCall, _status);                    \
		exit(-1);                                                              \
	}                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
	hipError_t _status = apiFuncCall;                                         \
	if (_status != hipSuccess) {                                              \
		fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
				__FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
		exit(-1);                                                              \
	}                                                                          \
} while (0)

#define CUPTI_CALL(call)                                                   \
  do {                                                                     \
	CUptiResult _status = call;                                            \
	if (_status != CUPTI_SUCCESS) {                                        \
	  const char *errstr;                                                  \
	  cuptiGetResultString(_status, &errstr);                              \
	  fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n", \
			  __FILE__, __LINE__, #call, errstr);                          \
	  if(_status == CUPTI_ERROR_LEGACY_PROFILER_NOT_SUPPORTED)             \
		  exit(0);                                                         \
	  else                                                                 \
		  exit(-1);                                                        \
	}                                                                      \
  } while (0)

#define ALIGN_SIZE (8)
#define ALIGN_BUFFER(buffer, align)                                            \
	(((uintptr_t) (buffer) & ((align)-1)) ? ((buffer) + (align) - ((uintptr_t) (buffer) & ((align)-1))) : (buffer))

// User data for event collection callback
typedef struct MetricData_st {
	// the device where metric is being collected
	hipDevice_t device;
	// the set of event groups to collect for a pass
	CUpti_EventGroupSet *eventGroups;
	// the current number of events collected in eventIdArray and
	// eventValueArray
	uint32_t eventIdx;
	// the number of entries in eventIdArray and eventValueArray
	uint32_t numEvents;
	// array of event ids
	CUpti_EventID *eventIdArray;
	// array of event values
	uint64_t *eventValueArray;
} MetricData_t;
MetricData_t metricData[PC_COUNT];

// Device code
__global__ void VecAdd(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

static void
initVec(int *vec, int n)
{
  for (int i=0; i< n; i++)
    vec[i] = i;
}

void CUPTIAPI
getMetricValueCallback(void *userdata, CUpti_CallbackDomain domain,
					   CUpti_CallbackId cbid, const CUpti_CallbackData *cbInfo)
{
  MetricData_t *metricData = (MetricData_t*)userdata;
  if(metricData->eventIdx >= metricData->numEvents){
	  metricData->eventIdx = 0;
  }

  unsigned int i, j, k;

  // This callback is enabled only for launch so we shouldn't see
  // anything else.
  if ((cbid != CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020) &&
      (cbid != CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000))
  {
    printf("%s:%d: unexpected cbid %d\n", __FILE__, __LINE__, cbid);
    exit(-1);
  }

  // on entry, enable all the event groups being collected this pass,
  // for metrics we collect for all instances of the event
  if (cbInfo->callbackSite == CUPTI_API_ENTER) {
	hipDeviceSynchronize();

	CUPTI_CALL(cuptiSetEventCollectionMode(cbInfo->context,
										   CUPTI_EVENT_COLLECTION_MODE_KERNEL));

	for (i = 0; i < metricData->eventGroups->numEventGroups; i++) {
	  uint32_t all = 1;
	  CUPTI_CALL(cuptiEventGroupSetAttribute(metricData->eventGroups->eventGroups[i],
											 CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES,
											 sizeof(all), &all));
	  CUPTI_CALL(cuptiEventGroupEnable(metricData->eventGroups->eventGroups[i]));
	}
  }

  // on exit, read and record event values
  if (cbInfo->callbackSite == CUPTI_API_EXIT) {
	hipDeviceSynchronize();

	// for each group, read the event values from the group and record
	// in metricData
	for (i = 0; i < metricData->eventGroups->numEventGroups; i++) {
	  CUpti_EventGroup group = metricData->eventGroups->eventGroups[i];
	  CUpti_EventDomainID groupDomain;
	  uint32_t numEvents, numInstances, numTotalInstances;
	  CUpti_EventID *eventIds;
	  size_t groupDomainSize = sizeof(groupDomain);
	  size_t numEventsSize = sizeof(numEvents);
	  size_t numInstancesSize = sizeof(numInstances);
	  size_t numTotalInstancesSize = sizeof(numTotalInstances);
	  uint64_t *values, normalized, *sum;
	  size_t valuesSize, eventIdsSize;
	  size_t numCountersRead = 0;

	  CUPTI_CALL(cuptiEventGroupGetAttribute(group,
											 CUPTI_EVENT_GROUP_ATTR_EVENT_DOMAIN_ID,
											 &groupDomainSize, &groupDomain));
	  CUPTI_CALL(cuptiDeviceGetEventDomainAttribute(metricData->device, groupDomain,
													CUPTI_EVENT_DOMAIN_ATTR_TOTAL_INSTANCE_COUNT,
													&numTotalInstancesSize, &numTotalInstances));
	  CUPTI_CALL(cuptiEventGroupGetAttribute(group,
											 CUPTI_EVENT_GROUP_ATTR_INSTANCE_COUNT,
											 &numInstancesSize, &numInstances));
	  CUPTI_CALL(cuptiEventGroupGetAttribute(group,
											 CUPTI_EVENT_GROUP_ATTR_NUM_EVENTS,
											 &numEventsSize, &numEvents));
	  eventIdsSize = numEvents * sizeof(CUpti_EventID);
	  eventIds = (CUpti_EventID *)malloc(eventIdsSize);
	  CUPTI_CALL(cuptiEventGroupGetAttribute(group,
											 CUPTI_EVENT_GROUP_ATTR_EVENTS,
											 &eventIdsSize, eventIds));

	  valuesSize = sizeof(uint64_t) * numInstances * numEvents;
	  values = (uint64_t *)malloc(valuesSize);

	  CUPTI_CALL(cuptiEventGroupReadAllEvents(group,
										  CUPTI_EVENT_READ_FLAG_NONE,
										  &valuesSize,
										  values,
										  &eventIdsSize,
										  eventIds,
										  &numCountersRead));

	  if (metricData->eventIdx >= metricData->numEvents) {
	  	printf("%d\n", metricData->eventIdx);
		fprintf(stderr, "error: too many events collected, metric expects only %d\n", (int)metricData->numEvents);
		exit(-1);
	  }

	  sum = (uint64_t *)calloc(sizeof(uint64_t), numEvents);
	  // sum collect event values from all instances
	  for (k = 0; k < numInstances; k++) {
		for (j = 0; j < numEvents; j++) {
			sum[j] += values[(k * numEvents) + j];
		}
	  }

	  for (j = 0; j < numEvents; j++) {
		// normalize the event value to represent the total number of
		// domain instances on the device
		normalized = (sum[j] * numTotalInstances) / numInstances;

		metricData->eventIdArray[metricData->eventIdx] = eventIds[j];
		metricData->eventValueArray[metricData->eventIdx] = normalized;
		metricData->eventIdx++;

		// print collected value
		{
		  char eventName[128];
		  size_t eventNameSize = sizeof(eventName) - 1;
		  CUPTI_CALL(cuptiEventGetAttribute(eventIds[j], CUPTI_EVENT_ATTR_NAME,
											&eventNameSize, eventName));
		  eventName[127] = '\0';
		 //  printf("\t%s = %llu (", eventName, (unsigned long long)sum[j]);
		 //  if (numInstances > 1) {
			// for (k = 0; k < numInstances; k++) {
			//   if (k != 0)
			// 	printf(", ");
			//   printf("%llu", (unsigned long long)values[(k * numEvents) + j]);
			// }
		 //  }

		  // printf(")\n");
		  // printf("\t%s (normalized) (%llu * %u) / %u = %llu\n",
				//  eventName, (unsigned long long)sum[j],
				//  numTotalInstances, numInstances,
				//  (unsigned long long)normalized);
		}
	  }

	  free(values);
	  free(sum);
	}

	for (i = 0; i < metricData->eventGroups->numEventGroups; i++)
	  CUPTI_CALL(cuptiEventGroupDisable(metricData->eventGroups->eventGroups[i]));
  }
}

static void
cleanUp(int *h_A, int *h_B, int *h_C, int *d_A, int *d_B, int *d_C)
{
  if (d_A)
    hipFree(d_A);
  if (d_B)
    hipFree(d_B);
  if (d_C)
    hipFree(d_C);

  // Free host memory
  if (h_A)
    free(h_A);
  if (h_B)
    free(h_B);
  if (h_C)
    free(h_C);
}

static void
runPass()
{
  size_t size = N * sizeof(int);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;
  int i, sum;

  // Allocate input vectors h_A and h_B in host memory
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);

  // Initialize input vectors
  initVec(h_A, N);
  initVec(h_B, N);
  memset(h_C, 0, size);

  // Allocate vectors in device memory
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel, let them occupy the least amount of space possible,
  // if the profiler has least memory footprint, it is easier to
  // detect performance counters of victim application
  threadsPerBlock = THREADS;
  blocksPerGrid = BLOCKS;
  // printf("Launching kernel: blocks %d, thread/block %d\n", blocksPerGrid, threadsPerBlock);

  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // Verify result
  for (i = 0; i < N; ++i) {
    sum = h_A[i] + h_B[i];
    if (h_C[i] != sum) {
      fprintf(stderr, "error: result verification failed\n");
      exit(-1);
    }
  }

  cleanUp(h_A, h_B, h_C, d_A, d_B, d_C);
}

static void CUPTIAPI
bufferRequested(uint8_t **buffer, size_t *size, size_t *maxNumRecords)
{
  uint8_t *rawBuffer;

  *size = 16 * 1024;
  rawBuffer = (uint8_t *)malloc(*size + ALIGN_SIZE);

  *buffer = ALIGN_BUFFER(rawBuffer, ALIGN_SIZE);
  *maxNumRecords = 0;

  if (*buffer == NULL) {
    printf("Error: out of memory\n");
    exit(-1);
  }
}

static void CUPTIAPI
bufferCompleted(hipCtx_t ctx, uint32_t streamId, uint8_t *buffer, size_t size, size_t validSize)
{
  CUpti_Activity *record = NULL;
  CUpti_ActivityKernel4 *kernel;

  //since we launched only 1 kernel, we should have only 1 kernel record
  CUPTI_CALL(cuptiActivityGetNextRecord(buffer, validSize, &record));

  kernel = (CUpti_ActivityKernel4 *)record;
  if (kernel->kind != CUPTI_ACTIVITY_KIND_KERNEL) {
    fprintf(stderr, "Error: expected kernel activity record, got %d\n", (int)kernel->kind);
    exit(-1);
  }

  kernelDuration = kernel->end - kernel->start;
  free(buffer);
}

////////////////////////////////////
//			MAIN FUNCTIONS		  //
////////////////////////////////////

void getMetric(char *metricName, int i, char *metric_data){

	// Subscribe //
	// setup launch callback for event collection //
	CUPTI_CALL(cuptiSubscribe(&subscriber[i], (CUpti_CallbackFunc)getMetricValueCallback, &metricData[i]));
	CUPTI_CALL(cuptiEnableCallback(1, subscriber[i], CUPTI_CB_DOMAIN_RUNTIME_API,
									CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020));
	CUPTI_CALL(cuptiEnableCallback(1, subscriber[i], CUPTI_CB_DOMAIN_RUNTIME_API,
									CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000));

	// allocate space to hold all the events needed for the metric
	CUPTI_CALL(cuptiMetricGetIdFromName(device, PC[i], &metricId[i]));
	CUPTI_CALL(cuptiMetricGetNumEvents(metricId[i], &metricData[i].numEvents));
	metricData[i].device = device;
	metricData[i].eventIdArray = (CUpti_EventID *)malloc(metricData[i].numEvents * sizeof(CUpti_EventID));
	metricData[i].eventValueArray = (uint64_t *)malloc(metricData[i].numEvents * sizeof(uint64_t));
	metricData[i].eventIdx = 0;

	// get the number of passes required to collect all the events
	// needed for the metric and the event groups for each pass
	CUPTI_CALL(cuptiMetricCreateEventGroupSets(context, sizeof(metricId[i]), &metricId[i], &passData[i]));
	for (unsigned int pass = 0; pass < passData[i]->numSets; pass++) {
		// printf("Pass %u\n", pass);
		metricData[i].eventGroups = passData[i]->sets + pass;
		runPass();
	}

	if (metricData[i].eventIdx != metricData[i].numEvents) {
		fprintf(stderr, "error: expected %u metric events, got %u\n",
		metricData[i].numEvents, metricData[i].eventIdx);
		exit(-1);
	}

	// use all the collected events to calculate the metric value
	CUPTI_CALL(cuptiMetricGetValue(device, metricId[i],
									metricData[i].numEvents * sizeof(CUpti_EventID),
									metricData[i].eventIdArray,
									metricData[i].numEvents * sizeof(uint64_t),
									metricData[i].eventValueArray,
									kernelDuration, &metricValue[i]));

	CUpti_MetricValueKind valueKind;
	size_t valueKindSize = sizeof(valueKind);
	CUPTI_CALL(cuptiMetricGetAttribute(metricId[i], CUPTI_METRIC_ATTR_VALUE_KIND, &valueKindSize, &valueKind));
	switch (valueKind) {
		case CUPTI_METRIC_VALUE_KIND_DOUBLE:
			sprintf(metric_data, "%f", metricValue[i].metricValueDouble);
			break;
		case CUPTI_METRIC_VALUE_KIND_UINT64:
			sprintf(metric_data, "%llu", (unsigned long long)metricValue[i].metricValueUint64);
			break;
		case CUPTI_METRIC_VALUE_KIND_INT64:
			sprintf(metric_data, "%lld", (long long)metricValue[i].metricValueInt64);
			break;
		case CUPTI_METRIC_VALUE_KIND_PERCENT:
			sprintf(metric_data, "%f", metricValue[i].metricValuePercent);
			break;
		case CUPTI_METRIC_VALUE_KIND_THROUGHPUT:
			sprintf(metric_data, "%llu", (unsigned long long)metricValue[i].metricValueThroughput);
			break;
		case CUPTI_METRIC_VALUE_KIND_UTILIZATION_LEVEL:
			sprintf(metric_data, "%u", (unsigned int)metricValue[i].metricValueUtilizationLevel);
			break;
		default:
			fprintf(stderr, "error: unknown value kind\n");
		exit(-1);
	}
	// Unsubscribe //
	CUPTI_CALL(cuptiUnsubscribe(subscriber[i]));
}

int collect_metrics(int count){

	// Open file to record metrics //
	ptr = fopen("result.csv","a");
	if(!ptr)
	{
		printf("file could not be opened\n");
		getchar();
		return -1;
	}

	// Memory Usage //
	hipMemGetInfo(&f, &t);
	sprintf(data, "%d,%lu,%lu,%lu", count, f, t, (t-f));

	// TODO: Collect other metrics //
	for(int i=0; i<PC_COUNT; i++){
		char *metric_data = (char *) malloc(100 * sizeof(char));
		getMetric(PC[i], i, metric_data);
		sprintf(data, "%s,%s", data, metric_data);
	}
	sprintf(data, "%s\n", data);

	printf("%s", data);
	fputs(data, ptr);
	fclose(ptr);
	return 0;
}

int timer()
{
	int count = 1;

	double time_counter = 0;

	clock_t this_time = clock();
	clock_t last_time = this_time;

	while(count < TIMES)
	{
		this_time = clock();

		time_counter += (double)(this_time - last_time);

		last_time = this_time;

		if(time_counter > (double)(NUM_SECONDS * CLOCKS_PER_SEC))
		{
			time_counter -= (double)(NUM_SECONDS * CLOCKS_PER_SEC);
			collect_metrics(count);
			count++;
		}
	}
	return 0;
}

int main(int argc, char *argv[]){

	// make sure activity is enabled before any CUDA API
	CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_KERNEL));

	DRIVER_API_CALL(hipInit(0));
	DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0) {
		printf("There is no device supporting CUDA.\n");
		return -2;
	}
	printf("CUDA Device Number: %d\n", deviceNum);

	DRIVER_API_CALL(hipDeviceGet(&device, deviceNum));
	DRIVER_API_CALL(hipDeviceGetName(deviceName, 32, device));
	printf("CUDA Device Name: %s\n", deviceName);

	DRIVER_API_CALL(hipCtxCreate(&context, 0, device));

	// need to collect duration of kernel execution without any event
	// collection enabled (some metrics need kernel duration as part of
	// calculation). The only accurate way to do this is by using the
	// activity API.
	{
		CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferRequested, bufferCompleted));
		runPass();
		hipDeviceSynchronize();
		CUPTI_CALL(cuptiActivityFlushAll(0));
	}

	// Set up output file //
	char header[5000];
	sprintf(header, "count,free,total,used");
	for(int i=0; i<PC_COUNT; i++){
		sprintf(header, "%s,%s", header, PC[i]);
	}
	sprintf(header, "%s\n", header);
	printf("%s", header);

	ptr = fopen("result.csv","w+");
	if(!ptr)
	{
		printf("file could not be opened\n");
		getchar();
		return -1;
	}
	fputs(header, ptr);
	fclose(ptr);

	// Start profiling //
	timer();
	
	return 0;
}
